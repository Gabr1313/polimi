
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>

#define ALPHABET_SIZE ('z' - 'a' + 1)

#define CHECK(call)                                                                       \
    do {                                                                                  \
        const hipError_t err = call;                                                     \
        if (err != hipSuccess) {                                                         \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(1);                                                                      \
        }                                                                                 \
    } while (0)

#define CHECK_KERNELCALL()                                                                \
    do {                                                                                  \
        const hipError_t err = hipGetLastError();                                       \
        if (err != hipSuccess) {                                                         \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(1);                                                                      \
        }                                                                                 \
    } while (0)

inline long long get_time() {
    struct timespec r;
    clock_gettime(0, &r);
    return r.tv_sec * (long long)1e9 + r.tv_nsec;
}

__global__ void histogramKernelSlow(const char *__restrict__ data,
                                    unsigned int *__restrict__ histogram, const int length) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < length) {
        int pos = (data[tid] | 0x20) - 'a';
        if (pos >= 0 && pos < ALPHABET_SIZE) atomicAdd(&histogram[pos], 1);
    }
}

__global__ void histogramKernel(const char *__restrict__ data, unsigned int *__restrict__ histogram,
                                const int length) {
    int tid    = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    __shared__ unsigned int histogramLocal[ALPHABET_SIZE];

#pragma unroll
    for (int i = threadIdx.x; i < ALPHABET_SIZE; i += blockDim.x) histogramLocal[i] = 0;
    __syncthreads();

    for (int i = tid; i < length; i += stride) {
        int pos = (data[i] | 0x20) - 'a';
        if (pos >= 0 && pos < ALPHABET_SIZE) atomicAdd(&histogramLocal[pos], 1);
    }
    __syncthreads();

#pragma unroll
    for (int i = threadIdx.x; i < ALPHABET_SIZE; i += blockDim.x) {
        unsigned int val = histogramLocal[i];
        if (val) atomicAdd(&histogram[i], val);
    }
}

void histogramSequential(char *data, unsigned int *histogram, int length) {
    for (int i = 0; i < length; i++) {
        int pos = (data[i] | 0x20) - 'a';  // lower case
        if (pos >= 0 && pos < ALPHABET_SIZE) histogram[pos]++;
    }
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("Usage: %s filename\n", argv[0]);
        exit(1);
    }

    FILE *fp = fopen(argv[1], "read");
    if (fp == NULL) exit(1);
    fseek(fp, 0L, SEEK_END);
    int   sz   = ftell(fp);
    char *text = (char *)malloc(sizeof(char) * sz);
    rewind(fp);
    fread(text, sizeof(char), sz, fp);
    fclose(fp);

    struct hipDeviceProp_t prop;
    CHECK(hipGetDeviceProperties(&prop, 0));
    int blockDim = prop.maxThreadsPerMultiProcessor;
    int gridDim  = prop.multiProcessorCount;
    // int blockDim = 32 * 1;
    // int gridDim  = (prop.maxThreadsPerMultiProcessor * prop.multiProcessorCount - 1) / blockDim + 1;

    unsigned int histogramCPU[ALPHABET_SIZE] = {0};
    unsigned int histogramGPU[ALPHABET_SIZE];
    unsigned int histogramGPU2[ALPHABET_SIZE];
    long long    start, end, elapsedCPU, elapsedGPU, elapsedGPU2;
    {
        start = get_time();
        histogramSequential(text, histogramCPU, sz);
        end        = get_time();
        elapsedCPU = end - start;
    }
    {
        start = get_time();
        char         *textCUDA;
        unsigned int *histogramCUDA;
        CHECK(hipMalloc(&textCUDA, sizeof(*textCUDA) * sz));
        CHECK(hipMemcpy(textCUDA, text, sizeof(*textCUDA) * sz, hipMemcpyHostToDevice));
        CHECK(hipMalloc(&histogramCUDA, sizeof(*histogramCUDA) * ALPHABET_SIZE));
        CHECK(hipMemset(histogramCUDA, 0, sizeof(*histogramCUDA) * ALPHABET_SIZE));
        dim3 blocksPerGrid((sz - 1) / blockDim + 1, 1, 1);
        dim3 threadsPerBlock(blockDim, 1, 1);

        histogramKernelSlow<<<blocksPerGrid, threadsPerBlock>>>(textCUDA, histogramCUDA, sz);
        CHECK_KERNELCALL();
        CHECK(hipDeviceSynchronize());
        CHECK(hipMemcpy(histogramGPU, histogramCUDA, ALPHABET_SIZE * sizeof(*histogramGPU),
                         hipMemcpyDeviceToHost));
        hipFree(textCUDA);
        hipFree(histogramCUDA);
        end        = get_time();
        elapsedGPU = end - start;
    }
    {
        start = get_time();
        char         *textCUDA;
        unsigned int *histogramCUDA;
        CHECK(hipMalloc(&textCUDA, sizeof(*textCUDA) * sz));
        CHECK(hipMemcpy(textCUDA, text, sizeof(*textCUDA) * sz, hipMemcpyHostToDevice));
        CHECK(hipMalloc(&histogramCUDA, sizeof(*histogramCUDA) * ALPHABET_SIZE));
        CHECK(hipMemset(histogramCUDA, 0, sizeof(*histogramCUDA) * ALPHABET_SIZE));
        dim3 blocksPerGrid(gridDim, 1, 1);
        dim3 threadsPerBlock(blockDim, 1, 1);
        histogramKernel<<<blocksPerGrid, threadsPerBlock>>>(textCUDA, histogramCUDA, sz);
        CHECK_KERNELCALL();
        CHECK(hipDeviceSynchronize());
        CHECK(hipMemcpy(histogramGPU2, histogramCUDA, ALPHABET_SIZE * sizeof(*histogramGPU),
                         hipMemcpyDeviceToHost));
        hipFree(textCUDA);
        hipFree(histogramCUDA);
        end         = get_time();
        elapsedGPU2 = end - start;
    }

    for (int i = 0; i < ALPHABET_SIZE; i++) {
        if (histogramCPU[i] != histogramGPU[i]) {
            printf("Result differs:\n");
            printf("CPU: ");
            for (int i = 0; i < ALPHABET_SIZE; i++) printf("%c:%d ", i + 'a', histogramCPU[i]);
            printf("\nGPU: ");
            for (int i = 0; i < ALPHABET_SIZE; i++) printf("%c:%d ", i + 'a', histogramGPU[i]);
            printf("\n");
            return 0;
        }
        if (histogramCPU[i] != histogramGPU2[i]) {
            printf("Result differs:\n");
            printf("CPU: ");
            for (int i = 0; i < ALPHABET_SIZE; i++) printf("%c:%d ", i + 'a', histogramCPU[i]);
            printf("\nGPU 2: ");
            for (int i = 0; i < ALPHABET_SIZE; i++) printf("%c:%d ", i + 'a', histogramGPU2[i]);
            printf("\n");
            return 0;
        }
    }

    printf("Time CPU     : %ld ns\n", elapsedCPU);
    printf("Time CUDA    : %ld ns\n", elapsedGPU);
    printf("Time CUDA2   : %ld ns\n", elapsedGPU2);
    printf("Ratio CPU/CUDA   : %.2f\n", (double)elapsedCPU / elapsedGPU);
    printf("Ratio CPU/CUDA2  : %.2f\n", (double)elapsedCPU / elapsedGPU2);

    free(text);

    return 0;
}
