
#include <hip/hip_runtime.h>
#include <assert.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

typedef int32_t  i32;
typedef uint32_t u32;
typedef uint64_t u64;
typedef float    f32;

// @todo decide when to call what funcion:
// - filter radius > 50          => run time error for saving input
// - filter radius in [2, 30]    => saving input is faster, otherwise slower
// - the most robust is the "no if" version -> a graph woulb be nice
// - saving filter is never faster (it is stays in cache by default)
// try iterate over filters (no I would not do this: in my mind it is useless)

u32 FILTER_RADIUS = 15;
u32 SIZE          = 500;
u32 LOOP_CPU      = 1;
u32 LOOP_GPU      = 100;

// better not to change
u32 BLOCK_DIM   = 8;  // becomes 64 because it is 2D
f32 EPS         = 0.00001;
u32 BLOCK_DIM_X = BLOCK_DIM;
u32 BLOCK_DIM_Y = BLOCK_DIM;

#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define MAX(a, b) ((a) > (b) ? (a) : (b))

u32 rng_seed() {
	struct timespec seed;
	clock_gettime(0, &seed);
	return (u32)(seed.tv_sec + seed.tv_nsec);
}

inline u64 get_time() {
	struct timespec r;
	clock_gettime(0, &r);
	return r.tv_sec * (u64)1e9 + r.tv_nsec;
}

void convolution_cpu(f32 *input, const f32 *filter, f32 *output, const u32 width, const u32 height,
                     const u32 filter_size, const u32 filter_radius) {
	assert(filter_size == filter_radius * 2 + 1);

	for (u32 outRow = 0; outRow < width; outRow++) {
		for (u32 outCol = 0; outCol < height; outCol++) {
			f32 value = 0.0f;
			for (u32 row = 0; row < filter_size; row++) {
				for (u32 col = 0; col < filter_size; col++) {
					i32 inRow = outRow - filter_radius + row;
					i32 inCol = outCol - filter_radius + col;
					if (inRow >= 0 && inRow < height && inCol >= 0 && inCol < width) {
						value += filter[col + row * filter_size] * input[inCol + inRow * width];
					}
				}
			}
			output[outCol + outRow * width] = value;
		}
	}

	/* for (u32 outRow = 0; outRow < width; outRow++) {
	    for (u32 outCol = 0; outCol < height; outCol++) {
	        f32 value     = 0.0f;
	        u32 row_start = MAX(0, (i32)(filter_radius - outRow));
	        u32 row_end   = MIN(filter_size, (i32)(height + filter_radius - outRow));
	        u32 col_start = MAX(0, (i32)(filter_radius - outCol));
	        u32 col_end   = MIN(filter_size, (i32)(width + filter_radius - outCol));
	        for (u32 row = row_start; row < row_end; row++) {
	            for (u32 col = col_start; col < col_end; col++) {
	                i32 inRow  = outRow - filter_radius + row;
	                i32 inCol  = outCol - filter_radius + col;
	                value     += filter[col + row * filter_size] * input[inCol + inRow * width];
	            }
	        }
	        output[outCol + outRow * width] = value;
	    }
	} */
}

__global__ void convolution_gpu_basic(f32 *input, const f32 *filter, f32 *output, const u32 width,
                                      const u32 height, const u32 filter_size,
                                      const u32 filter_radius) {
	const u32 outCol = blockIdx.x * blockDim.x + threadIdx.x;
	const u32 outRow = blockIdx.y * blockDim.y + threadIdx.y;
	if (outCol < width && outRow < height) {
		f32 value = 0.0f;
		for (u32 row = 0; row < filter_size; row++) {
			for (u32 col = 0; col < filter_size; col++) {
				i32 inRow = outRow - filter_radius + row;
				i32 inCol = outCol - filter_radius + col;
				if (inRow >= 0 && inRow < height && inCol >= 0 && inCol < width) {
					value += filter[col + row * filter_size] * input[inCol + inRow * width];
				}
			}
		}
		output[outRow * width + outCol] = value;
	}
}

__global__ void convolution_gpu_no_if(f32 *input, const f32 *filter, f32 *output, const u32 width,
                                      const u32 height, const u32 filter_size,
                                      const u32 filter_radius) {
	const u32 outCol = blockIdx.x * blockDim.x + threadIdx.x;
	const u32 outRow = blockIdx.y * blockDim.y + threadIdx.y;
	if (outCol < width && outRow < height) {
		f32 value     = 0.0f;
		u32 row_start = MAX(0, (i32)(filter_radius - outRow));
		u32 row_end   = MIN(filter_size, (i32)(height + filter_radius - outRow));
		u32 col_start = MAX(0, (i32)(filter_radius - outCol));
		u32 col_end   = MIN(filter_size, (i32)(width + filter_radius - outCol));
		for (u32 row = row_start; row < row_end; row++) {
			for (u32 col = col_start; col < col_end; col++) {
				i32 inRow  = outRow - filter_radius + row;
				i32 inCol  = outCol - filter_radius + col;
				value     += filter[col + row * filter_size] * input[inCol + inRow * width];
			}
		}
		output[outCol + outRow * width] = value;
	}
}

__global__ void convolution_gpu_shared_filter(f32 *input, const f32 *filter, f32 *output,
                                              const u32 width, const u32 height,
                                              const u32 filter_size, const u32 filter_radius) {
	const u32 outCol = blockIdx.x * blockDim.x + threadIdx.x;
	const u32 outRow = blockIdx.y * blockDim.y + threadIdx.y;

	extern __shared__ f32 local_filter[];
	for (u32 i = threadIdx.y; i < filter_size; i += blockDim.y) {
		for (u32 j = threadIdx.x; j < filter_size; j += blockDim.x) {
			local_filter[i * filter_size + j] = filter[i * filter_size + j];
		}
	}
	__syncthreads();

	if (outCol < width && outRow < height) {
		f32 value     = 0.0f;
		u32 row_start = MAX(0, (i32)(filter_radius - outRow));
		u32 row_end   = MIN(filter_size, (i32)(height + filter_radius - outRow));
		u32 col_start = MAX(0, (i32)(filter_radius - outCol));
		u32 col_end   = MIN(filter_size, (i32)(width + filter_radius - outCol));
		for (u32 row = row_start; row < row_end; row++) {
			for (u32 col = col_start; col < col_end; col++) {
				i32 inRow  = outRow - filter_radius + row;
				i32 inCol  = outCol - filter_radius + col;
				value     += local_filter[row * filter_size + col] * input[inRow * width + inCol];
			}
		}
		output[outCol + outRow * width] = value;
	}
}

__global__ void convolution_gpu_shared_input(f32 *input, const f32 *filter, f32 *output,
                                             const u32 width, const u32 height,
                                             const u32 filter_size, const u32 filter_radius) {
	const u32 outCol = blockIdx.x * blockDim.x + threadIdx.x;
	const u32 outRow = blockIdx.y * blockDim.y + threadIdx.y;

	extern __shared__ f32 local_input[];

	u32 local_input_x_size = blockDim.x + 2 * filter_radius;
	u32 local_input_y_size = blockDim.y + 2 * filter_radius;
	for (u32 i = threadIdx.x; i < local_input_x_size; i += blockDim.x) {
		for (u32 j = threadIdx.y; j < local_input_y_size; j += blockDim.y) {
			i32 inCol = blockIdx.x * blockDim.x + i - filter_radius;
			i32 inRow = blockIdx.y * blockDim.y + j - filter_radius;
			if (inCol >= 0 && inCol < width && inRow >= 0 && inRow < height) {
				local_input[i + j * local_input_x_size] = input[inCol + inRow * width];
			}
		}
	}
	__syncthreads();

	if (outCol < width && outRow < height) {
		f32 value     = 0.0f;
		u32 row_start = MAX(0, (i32)(filter_radius - outRow));
		u32 row_end   = MIN(filter_size, (i32)(height + filter_radius - outRow));
		u32 col_start = MAX(0, (i32)(filter_radius - outCol));
		u32 col_end   = MIN(filter_size, (i32)(width + filter_radius - outCol));
		for (u32 row = row_start; row < row_end; row++) {
			for (u32 col = col_start; col < col_end; col++) {
				i32 inCol  = threadIdx.x + col;
				i32 inRow  = threadIdx.y + row;
				value     += filter[col + row * filter_size] *
				         local_input[inCol + inRow * local_input_x_size];
			}
		}
		output[outCol + outRow * width] = value;
	}
}

#define CHECK(call)                                                                       \
	do {                                                                                  \
		const hipError_t err = call;                                                     \
		if (err != hipSuccess) {                                                         \
			printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
			exit(1);                                                                      \
		}                                                                                 \
	} while (0)

#define CHECK_KERNELCALL()                                                                \
	do {                                                                                  \
		const hipError_t err = hipGetLastError();                                       \
		if (err != hipSuccess) {                                                         \
			printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
			exit(1);                                                                      \
		}                                                                                 \
	} while (0)

#define CALL_CPU(func, elapsed, output, input, filter, width, height, filter_size, filter_radius) \
	do {                                                                                          \
		u64 start = get_time();                                                                   \
		for (int i = 0; i < LOOP_CPU; i++) {                                                      \
			func(input, filter, output, width, height, filter_size, filter_radius);               \
		}                                                                                         \
		u64 end = get_time();                                                                     \
		elapsed = (end - start) / LOOP_CPU;                                                       \
	} while (0)

#define CALL_KERNELL(func, shared_mem, elapsed, output, input, filter, width, height, filter_size, \
                     filter_radius)                                                                \
	do {                                                                                           \
		f32 *input_cuda, *output_cuda;                                                             \
		f32 *filter_cuda;                                                                          \
		CHECK(hipMalloc(&input_cuda, sizeof(*input_cuda) * width * height));                      \
		CHECK(hipMemcpy(input_cuda, input, sizeof(*input_cuda) * width * height,                  \
		                 hipMemcpyHostToDevice));                                                 \
		CHECK(hipMalloc(&filter_cuda, sizeof(*filter_cuda) * filter_size * filter_size));         \
		CHECK(hipMemcpy(filter_cuda, filter, sizeof(*filter_cuda) * filter_size * filter_size,    \
		                 hipMemcpyHostToDevice));                                                 \
		CHECK(hipMalloc(&output_cuda, sizeof(*output_cuda) * width * height));                    \
                                                                                                   \
		u32  gridDimX = (width - 1) / BLOCK_DIM_X + 1;                                             \
		u32  gridDimY = (height - 1) / BLOCK_DIM_Y + 1;                                            \
		dim3 blocksPerGrid(gridDimX, gridDimY, 1);                                                 \
		dim3 threadsPerBlock(BLOCK_DIM_X, BLOCK_DIM_Y, 1);                                         \
                                                                                                   \
		u64 start = get_time();                                                                    \
		for (int i = 0; i < LOOP_GPU; i++) {                                                       \
			func<<<blocksPerGrid, threadsPerBlock, shared_mem>>>(                                  \
			    input_cuda, filter_cuda, output_cuda, width, height, filter_size, filter_radius);  \
			CHECK_KERNELCALL();                                                                    \
			CHECK(hipDeviceSynchronize());                                                        \
		}                                                                                          \
		u64 end = get_time();                                                                      \
		elapsed = (end - start) / LOOP_GPU;                                                        \
                                                                                                   \
		CHECK(hipMemcpy(output, output_cuda, sizeof(*output) * width * height,                    \
		                 hipMemcpyDeviceToHost));                                                 \
                                                                                                   \
		CHECK(hipFree(output_cuda));                                                              \
		CHECK(hipFree(input_cuda));                                                               \
		CHECK(hipFree(filter_cuda));                                                              \
	} while (0)

#define CHECK_EQUAL_RESULT(to_print, output_cpu, output_gpu, i, width, height, seed)             \
	do {                                                                                         \
		if (fabs(output_cpu[i] - output_gpu[i]) > EPS) {                                         \
			fprintf(stderr,                                                                      \
			        to_print                                                                     \
			        "Output differs (index = %u (%u %u), width = %u, height = %u, seed = %u)"    \
			        ": %f != %f\n",                                                              \
			        i, i / width, i % width, width, height, seed, output_cpu[i], output_gpu[i]); \
			exit(1);                                                                             \
		}                                                                                        \
	} while (0)

i32 main(i32 argc, char **argv) {
	// if (argc != 2) {
	// 	printf("Please specify matrix dimensions\n");
	// exit(1);
	// }
	// const u32 size   = atoi(argv[1]);
	const u32 width = SIZE, height = SIZE;
	const u32 filter_radius = FILTER_RADIUS;
	const u32 filter_size   = filter_radius * 2 + 1;

	f32 *input            = (f32 *)malloc(sizeof(*input) * width * height);
	f32 *filter           = (f32 *)malloc(sizeof(*filter) * filter_size * filter_size);
	f32 *output_cpu       = (f32 *)malloc(sizeof(*output_cpu) * width * height);
	f32 *output_gpu_basic = (f32 *)malloc(sizeof(*output_gpu_basic) * width * height);
	f32 *output_gpu_no_if = (f32 *)malloc(sizeof(*output_gpu_no_if) * width * height);
	f32 *output_gpu_shared_filter =
	    (f32 *)malloc(sizeof(*output_gpu_shared_filter) * width * height);
	f32 *output_gpu_shared_input = (f32 *)malloc(sizeof(*output_gpu_shared_input) * width * height);

	u32 seed = rng_seed();
	srand(seed);
	for (u32 i = 0; i < filter_size * filter_size; i++) filter[i] = (f32)rand() / RAND_MAX;
	for (u32 i = 0; i < width * height; i++) input[i] = (f32)rand() / RAND_MAX;

	u64 elapsed_cpu, elapsed_gpu_basic, elapsed_gpu_shared_filter, elapsed_gpu_no_if,
	    elapsed_gpu_shared_input;

	CALL_CPU(convolution_cpu, elapsed_cpu, output_cpu, input, filter, width, height, filter_size,
	         filter_radius);
	CALL_KERNELL(convolution_gpu_basic, 0, elapsed_gpu_basic, output_gpu_basic, input, filter,
	             width, height, filter_size, filter_radius);
	CALL_KERNELL(convolution_gpu_no_if, 0, elapsed_gpu_no_if, output_gpu_no_if, input, filter,
	             width, height, filter_size, filter_radius);
	CALL_KERNELL(convolution_gpu_shared_filter, sizeof(*filter_cuda) * filter_size * filter_size,
	             elapsed_gpu_shared_filter, output_gpu_shared_filter, input, filter, width, height,
	             filter_size, filter_radius);
	CALL_KERNELL(
	    convolution_gpu_shared_input,
	    sizeof(*input_cuda) * (BLOCK_DIM_X + 2 * filter_radius) * (BLOCK_DIM_Y + 2 * filter_radius),
	    elapsed_gpu_shared_input, output_gpu_shared_input, input, filter, width, height,
	    filter_size, filter_radius);

	for (u32 i = 0; i < width * height; i++) {
		CHECK_EQUAL_RESULT("Basic: ", output_cpu, output_gpu_basic, i, width, height, seed);
		CHECK_EQUAL_RESULT("No if: ", output_cpu, output_gpu_no_if, i, width, height, seed);
		CHECK_EQUAL_RESULT("Shared filter: ", output_cpu, output_gpu_shared_filter, i, width,
		                   height, seed);
		CHECK_EQUAL_RESULT("Shared input:", output_cpu, output_gpu_shared_input, i, width, height,
		                   seed);
	}

	printf("Time CPU               : %lu ns\n", elapsed_cpu);
	printf("Time CUDA basic        : %lu ns (ratio: %lu)\n", elapsed_gpu_basic,
	       elapsed_cpu / elapsed_gpu_basic);
	printf("Time CUDA no if        : %lu ns (ratio: %lu)\n", elapsed_gpu_no_if,
	       elapsed_cpu / elapsed_gpu_no_if);
	printf("Time CUDA shared filter: %lu ns (ratio: %lu)\n", elapsed_gpu_shared_filter,
	       elapsed_cpu / elapsed_gpu_shared_filter);
	printf("Time CUDA shared input : %lu ns (ratio: %lu)\n", elapsed_gpu_shared_input,
	       elapsed_cpu / elapsed_gpu_shared_input);

	free(input);
	free(filter);
	free(output_cpu);
	free(output_gpu_basic);
	free(output_gpu_no_if);
	free(output_gpu_shared_input);
	free(output_gpu_shared_filter);

	return 0;
}
